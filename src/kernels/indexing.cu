#include "hip/hip_runtime.h"
#include "compatibility.cuh"
#include<stdint.h>

#define SCATTER_OP(TYPENAME, INDEX_TYPENAME, FN_NAME, OP) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { \
    const size_t numel = left_size * right_size;\
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {\
          const size_t pre = i / right_size;\
          const size_t post = i % right_size;\
          for (unsigned int j = 0; j < src_dim_size; ++j) {\
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;\
              const size_t idx = ids[src_i];\
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;\
              out[dst_i] OP inp[src_i];\
          }\
      }\
 } \

#if __CUDA_ARCH__ >= 800
SCATTER_OP(__hip_bfloat16, int64_t, scatter_i64_bf16, =)
SCATTER_OP(__hip_bfloat16, uint32_t, scatter_u32_bf16, =)
SCATTER_OP(__hip_bfloat16, uint8_t, scatter_u8_bf16, =)
#endif

#if __CUDA_ARCH__ >= 530
SCATTER_OP(__half, uint32_t, scatter_u32_f16, =)
SCATTER_OP(__half, uint8_t, scatter_u8_f16, =)
#endif


#pragma region scatter_assign
SCATTER_OP(float, int64_t, scatter_i64_f32, =)
SCATTER_OP(double, int64_t, scatter_i64_f64, =)
SCATTER_OP(uint8_t, int64_t, scatter_i64_u8, =)
SCATTER_OP(int64_t, int64_t, scatter_i64_i64, =)
SCATTER_OP(uint32_t, int64_t, scatter_i64_u32, =)

SCATTER_OP(float, uint32_t, scatter_u32_f32, =)
SCATTER_OP(double, uint32_t, scatter_u32_f64, =)
SCATTER_OP(uint8_t, uint32_t, scatter_u32_u8, =)
SCATTER_OP(int64_t, uint32_t, scatter_u32_i64, =)
SCATTER_OP(uint32_t, uint32_t, scatter_u32_u32, =)

SCATTER_OP(float, uint8_t, scatter_u8_f32, =)
SCATTER_OP(double, uint8_t, scatter_u8_f64, =)
SCATTER_OP(uint8_t, uint8_t, scatter_u8_u8, =)
SCATTER_OP(uint32_t, uint8_t, scatter_u8_u32, =)
SCATTER_OP(int64_t, uint8_t, scatter_u8_i64, =)
#pragma endregion scatter_assign